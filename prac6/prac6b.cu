//
// include files
//


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

//
// template kernel routine
// 

template  <class T>
__global__ void my_first_kernel(T *x)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  x[tid] = threadIdx.x;
}


//
// CUDA routine to be called by main code
//

extern
int prac6(int nblocks, int nthreads)
{
  float *h_x, *d_x;
  int   *h_i, *d_i;
  double *h_d, *d_d;
  int   nsize, n; 

  // allocate memory for arrays

  nsize = nblocks*nthreads ;

  h_x = (float *)malloc(nsize*sizeof(float));
  hipMalloc((void **)&d_x, nsize*sizeof(float));

  h_i = (int *)malloc(nsize*sizeof(int));
  hipMalloc((void **)&d_i, nsize*sizeof(int));

  h_d = (double *)malloc(nsize*sizeof(double));
  hipMalloc((void **)&d_d, nsize*sizeof(double));

  // execute kernel for float

  my_first_kernel<<<nblocks,nthreads>>>(d_x);
  hipMemcpy(h_x,d_x,nsize*sizeof(float),hipMemcpyDeviceToHost);
  for (n=0; n<nsize; n++) printf(" n,  x  =  %d  %.9g \n",n,h_x[n]);

  // execute kernel for ints

  my_first_kernel<<<nblocks,nthreads>>>(d_i);
  hipMemcpy(h_i,d_i,nsize*sizeof(int),hipMemcpyDeviceToHost);
  for (n=0; n<nsize; n++) printf(" n,  i  =  %d  %d \n",n,h_i[n]);

  // execute kernal for double

  my_first_kernel<<<nblocks,nthreads>>>(d_d);
  hipMemcpy(h_d,d_d,nsize*sizeof(double),hipMemcpyDeviceToHost);
  for (n=0; n<nsize; n++) printf(" n,  i  =  %d  %.17g \n",n,h_d[n]);

  // free memory 

  hipFree(d_x);
  free(h_x);
  hipFree(d_i);
  free(h_i);
  hipFree(d_d);
  free(h_d);

  return 0;
}

 
